#include "../include/cudaTools.cuh"

size_t getDeviceMem(int device)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    return props.totalGlobalMem;
}
