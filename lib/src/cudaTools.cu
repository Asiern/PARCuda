#include "../include/cudaTools.cuh"
#include <stdlib.h>

size_t getDeviceMem(int device)
{
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    return props.totalGlobalMem;
}
