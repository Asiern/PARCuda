#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <iostream>

#define n_threads 1024

__global__ void add_kernel(float *A, float *B, float *out)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    out[x] = A[x] + B[x];
}
__global__ void mul_kernel(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = pos % b;
    unsigned int col = pos / b;

    if (row < a && col < b)
    {
        float sum = 0;
        for (int i = 0; i < a; i++)
            sum += A[row * a + i] * B[i * a + col];
        out[row * a + col] = sum;
    }
}

int matrix_add_cuda(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    if (a != x || b != y)
        return 1;

#ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    size_t size = sizeof(float) * a * b;

    // Allocate Memory
    float *d_A;
    hipMalloc(&d_A, size);
    if (hipMemcpy(d_A, A, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << "Error no se puede reservar memoria (Mat add)" << std::endl;
        return 1;
    }

    float *d_B;
    hipMalloc(&d_B, size);
    if (hipMemcpy(d_B, B, size, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << "Error al copiar matriz a memoria (Mat add)" << std::endl;
        return 1;
    }

    float *d_out;
    if (hipMalloc(&d_out, size) != hipSuccess)
    {
        std::cout << "Error no se puede reservar memoria (Mat add)" << std::endl;
        return 1;
    }

    // Call Kernel
#ifdef DEBUG
    hipEventRecord(start);
#endif
    add_kernel<<<32, n_threads>>>(d_A, d_B, d_out);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda add kernel ex time(ms): " << time_ms << std::endl;
#endif

    // Copy results
    if (hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost) != hipSuccess)
        return 1;

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
    return 0;
}

int matrix_mul_cuda(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    // Return if matrix dimensions not compatible
    if (b != x)
        return 1;

#ifdef DEBUG
    // Timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    // Allocate Memory

    // A Matrix
    float *d_A;
    size_t sizeA = sizeof(float) * a * b;
    if (hipMalloc(&d_A, sizeA) != hipSuccess)
    {
        std::cout << "Error no se puede reservar memoria" << std::endl;
        return 1;
    }
    if (hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice) != hipSuccess)
    {
        std::cout << "Error al copiar matriz a memoria" << std::endl;
        return 1;
    }

    // B Matrix
    float *d_B;
    size_t sizeB = sizeof(float) * x * y;
    if (hipMalloc(&d_B, sizeB) == hipErrorOutOfMemory)
    {
        std::cout << "Error no se puede reservar memoria" << std::endl;
        return 1;
    }
    if (hipMemcpy(d_B, B, sizeA, hipMemcpyHostToDevice) > 0)
    {
        std::cout << "Error al copiar matriz a memoria" << std::endl;
        return 1;
    }

    // Out Matrix
    float *d_out;
    size_t sizeOut = sizeof(float) * a * y;
    if (hipMalloc(&d_out, sizeOut) == hipErrorOutOfMemory)
    {
        std::cout << "Error no se puede reservar memoria" << std::endl;
        return 1;
    }

    // Set grid dimensions
    dim3 n_blocks = dim3((a * b / n_threads) + 1);

#ifdef DEBUG
    // Start timer
    hipEventRecord(start);
#endif
    // Call kernel
    mul_kernel<<<n_blocks, n_threads>>>(d_A, d_B, d_out, a, b, x, y);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda mul kernel ex time(ms): " << time_ms << std::endl;
#endif

    // Copy results
    hipMemcpy(out, d_out, sizeOut, hipMemcpyDeviceToHost);

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);

    return 0;
}

int matrix_mul_add_cuda(float *A, float *B, float *C, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y, unsigned int p, unsigned int q)
{
    float *mul = (float *)malloc(sizeof(float) * a * y);
    if (matrix_mul_cuda(A, B, mul, a, b, x, y))
    {
        free(mul);
        return 1;
    }
    if (matrix_add_cuda(C, mul, out, p, q, a, y))
    {
        free(mul);
        return 1;
    }
    free(mul);
    return 0;
}
