#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <iostream>

#define n_threads 1024

__global__ void add_kernel(float *A, float *B, float *out)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    out[x] = A[x] + B[x];
}
__global__ void mul_kernel(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    int row = blockIdx.y * a + threadIdx.y;
    int col = blockIdx.x * a + threadIdx.x;

    if (row < a && col < a)
    {
        float sum = 0;
        for (int i = 0; i < a; i++)
            sum += A[row * a + i] * B[i * a + col];
        out[row * a + col] = sum;
    }
}

int matrix_add_cuda(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    if (a != x || b != y)
        return 1;

#ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    size_t size = sizeof(float) * a * b;

    // Allocate Memory
    float *d_A;
    hipMalloc(&d_A, size);
    if (hipMemcpy(d_A, A, size, hipMemcpyHostToDevice) != hipSuccess)
        return 1;

    float *d_B;
    hipMalloc(&d_B, size);
    if (hipMemcpy(d_B, B, size, hipMemcpyHostToDevice) != hipSuccess)
        return 1;

    float *d_out;
    hipMalloc(&d_out, size);

    // Call Kernel
#ifdef DEBUG
    hipEventRecord(start);
#endif
    add_kernel<<<32, n_threads>>>(d_A, d_B, d_out);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda add kernel ex time(ms): " << time_ms << std::endl;
#endif

    // Copy results
    if (hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost) != hipSuccess)
        return 1;

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);
    return 0;
}

int matrix_mul_cuda(float *A, float *B, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y)
{
    if (b != x)
        return 1;

#ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    // Allocate Memory
    float *d_A;
    size_t sizeA = sizeof(float) * a * b;
    hipMalloc(&d_A, sizeA);
    hipMemcpy(d_A, A, sizeA, hipMemcpyHostToDevice);
    float *d_B;
    size_t sizeB = sizeof(float) * x * y;
    hipMalloc(&d_B, sizeB);
    hipMemcpy(d_B, B, sizeB, hipMemcpyHostToDevice);
    float *d_out;
    size_t sizeOut = sizeof(float) * a * y;
    hipMalloc(&d_out, sizeOut);

    dim3 n_blocks = dim3(a * b / n_threads);

#ifdef DEBUG
    hipEventRecord(start);
#endif
    // Call kernel
    mul_kernel<<<n_blocks, n_threads>>>(d_A, d_B, d_out, a, b, x, y);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda mul kernel ex time(ms): " << time_ms << std::endl;
#endif

    // Copy results
    hipMemcpy(out, d_out, sizeOut, hipMemcpyDeviceToHost);

    // Free
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_out);

    return 0;
}

int matrix_mul_add_cuda(float *A, float *B, float *C, float *out, unsigned int a, unsigned int b, unsigned int x, unsigned int y, unsigned int p, unsigned int q)
{
    float *mul = (float *)malloc(sizeof(float) * a * y);
    if (matrix_mul_cuda(A, B, mul, a, b, x, y))
    {
        free(mul);
        return 1;
    }
    if (matrix_add_cuda(C, mul, out, p, q, a, y))
    {
        free(mul);
        return 1;
    }
    free(mul);
    return 0;
}
