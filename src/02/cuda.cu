#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <iostream>

#define NTHREADS 1024

__global__ void transpose_kernel(float *in, float *out, int n, int m)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x > n * m)
        return;
    unsigned int fila = x % m;
    unsigned int columna = x / m;
    unsigned int y = fila * n + columna;
    out[x] = in[y];
}

void transpose_cuda(float *C, unsigned int N, unsigned int M)
{

#ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    // Create memory copies
    size_t size = sizeof(float) * N * M;
    float *in, *out;
    hipMalloc(&in, size);
    hipMalloc(&out, size);
    hipMemcpy(in, C, size, hipMemcpyHostToDevice);

#ifdef DEBUG
    hipEventRecord(start);
#endif
    dim3 nblocks((N * M / NTHREADS) + 1);
    // Launch Kernel
    transpose_kernel<<<nblocks, NTHREADS>>>(in, out, N, M);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda transpose kernel ex time(ms): " << time_ms << std::endl;
#endif

    // Save results
    hipMemcpy(C, out, size, hipMemcpyDeviceToHost);

    // Free
    hipFree(in);
    hipFree(out);
}
