#include "hip/hip_runtime.h"
#include "cuda.cuh"
#include <iostream>

__global__ void transpose_kernel(float *in, float *out, int n, int m)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int fila = x % m;
    unsigned int columna = x / m;
    unsigned int y = fila * n + columna;
    out[x] = in[y];
}

void transpose_cuda(float *C, unsigned int N, unsigned int M)
{

#ifdef DEBUG
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    // Create memory copies
    size_t size = sizeof(float) * N * M;
    float *in, *out;
    hipMalloc(&in, size);
    hipMalloc(&out, size);
    hipMemcpy(in, C, size, hipMemcpyHostToDevice);

#ifdef DEBUG
    hipEventRecord(start);
#endif
    // Launch Kernel
    transpose_kernel<<<32, 32>>>(in, out, N, M);
    hipDeviceSynchronize();
#ifdef DEBUG
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float time_ms = 0;
    hipEventElapsedTime(&time_ms, start, stop);
    std::cout << "Cuda mul kernel ex time(ns): " << time_ms * 1000000 << std::endl;
#endif

    // Save results
    hipMemcpy(C, out, size, hipMemcpyDeviceToHost);

    // Free
    hipFree(in);
    hipFree(out);
}
